#include "hip/hip_runtime.h"
#include "cudafuncs.cuh"
#include "Marvin3DMatch.cuh"

// CUDA kernal function to compute TDF voxel grid values given a point cloud (warning: approximate, but fast)
__global__ void computeTDFKernel(int loopIDX, float * voxelGridOcc, float * voxelGridTDF,
								 int voxelGridDimX, int voxelGridDimY, int voxelGridDimZ,
								 float voxelSize, float truncMargin)
{
	int voxelIDX = loopIDX * CUDA_NUM_THREADS * CUDA_MAX_NUM_BLOCKS + blockIdx.x * CUDA_NUM_THREADS + threadIdx.x;
	if (voxelIDX >= (voxelGridDimX * voxelGridDimY * voxelGridDimZ))
		return;
	int pointGridZ = (int)floor((float)voxelIDX / ((float)voxelGridDimX * (float)voxelGridDimY));
	int pointGridY = (int)floor(((float)voxelIDX - ((float)pointGridZ * (float)voxelGridDimX * (float)voxelGridDimY)) / (float)voxelGridDimX);
	int pointGridX = (int)((float)voxelIDX - ((float)pointGridZ * (float)voxelGridDimX * (float)voxelGridDimY) - ((float)pointGridY * (float)voxelGridDimX));

	int searchRadius = (int)round(truncMargin / voxelSize);

	if (voxelGridOcc[voxelIDX] > 0)
	{
		voxelGridTDF[voxelIDX] = 1.0f;	// on surface
		return;
	}

	//Find closest surface point
	for (int i = max(0, pointGridX - searchRadius); i < min(voxelGridDimX, pointGridX + searchRadius + 1); ++i)
	{
		for (int j = max(0, pointGridY - searchRadius); j < min(voxelGridDimY, pointGridY + searchRadius + 1); ++j)
		{
			for (int k = max(0, pointGridZ - searchRadius); k < min(voxelGridDimZ, pointGridZ + searchRadius + 1); ++k)
			{
				int idx = k * voxelGridDimX * voxelGridDimY + j * voxelGridDimX + i;
				if (voxelGridOcc[idx] > 0)
				{
					float xd = (float)(pointGridX - i);
					float yd = (float)(pointGridY - j);
					float zd = (float)(pointGridZ - k);
					float dist = sqrtf(xd * xd + yd * yd + zd * zd) / (float)searchRadius;
					if ((1.0f - dist) > voxelGridTDF[voxelIDX])
						voxelGridTDF[voxelIDX] = 1.0f - dist;
				}
			}
		}
	}
}

__global__ void computeTDFKernelSun(int loopIDX, float * voxelGridOcc, float * voxelGridTDF,
								 int voxelGridDimX, int voxelGridDimY, int voxelGridDimZ,
								 float voxelSize, float truncMargin,int modelNum)
{
	int voxelIDX = loopIDX * CUDA_NUM_THREADS * CUDA_MAX_NUM_BLOCKS + blockIdx.x * CUDA_NUM_THREADS + threadIdx.x;
	if (voxelIDX >= (voxelGridDimX * voxelGridDimY * voxelGridDimZ))
		return;
	int pointGridZ = (int)floor((float)voxelIDX / ((float)voxelGridDimX * (float)voxelGridDimY));
	int pointGridY = (int)floor(((float)voxelIDX - ((float)pointGridZ * (float)voxelGridDimX * (float)voxelGridDimY)) / (float)voxelGridDimX);
	int pointGridX = (int)((float)voxelIDX - ((float)pointGridZ * (float)voxelGridDimX * (float)voxelGridDimY) - ((float)pointGridY * (float)voxelGridDimX));

	int searchRadius = (int)round(truncMargin / voxelSize);
	int voxelGirdSize = voxelGridDimX * voxelGridDimY * voxelGridDimZ;

	for (int l = 0; l < modelNum; ++l)
	{
		if (voxelGridOcc[voxelIDX + l * voxelGirdSize] > 0 && voxelGridOcc[voxelIDX + l * voxelGirdSize] < 1)
		{
			voxelGridTDF[voxelIDX + l * voxelGirdSize] = 1.0f;	// on surface
			continue;
		}

		//Find closest surface point
		for (int i = max(0, pointGridX - searchRadius); i < min(voxelGridDimX, pointGridX + searchRadius + 1); ++i)
		{
			for (int j = max(0, pointGridY - searchRadius); j < min(voxelGridDimY, pointGridY + searchRadius + 1); ++j)
			{
				for (int k = max(0, pointGridZ - searchRadius); k < min(voxelGridDimZ, pointGridZ + searchRadius + 1); ++k)
				{
					int idx = k * voxelGridDimX * voxelGridDimY + j * voxelGridDimX + i + l * voxelGirdSize;
					if (voxelGridOcc[idx] > 0 && voxelGridOcc[idx] < 1)
					{
						float xd = (float)(pointGridX - i);
						float yd = (float)(pointGridY - j);
						float zd = (float)(pointGridZ - k);
						float dist = sqrtf(xd * xd + yd * yd + zd * zd) / (float)searchRadius;
						if ((1.0f - dist) > voxelGridTDF[voxelIDX + l * voxelGirdSize])
							voxelGridTDF[voxelIDX + l * voxelGirdSize] = 1.0f - dist;
					}
				}
			}
		}
	}
}

void computeTDFVoxelGrid(float * voxelGridOcc,
						 float * voxelGridTDF,
						 int voxelGridDimX,
						 int voxelGridDimY,
						 int voxelGridDimZ,
						 float voxelSize,
						 float truncMargin)
{
	float * d_voxelGridOcc;
	float * d_voxelGridTDF;
	hipMalloc(&d_voxelGridOcc, voxelGridDimX * voxelGridDimY * voxelGridDimZ * sizeof(float));
	hipMalloc(&d_voxelGridTDF, voxelGridDimX * voxelGridDimY * voxelGridDimZ * sizeof(float));
	marvin::checkCUDA(__LINE__, hipGetLastError());
	hipMemcpy(d_voxelGridOcc, voxelGridOcc, voxelGridDimX * voxelGridDimY * voxelGridDimZ * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_voxelGridTDF, voxelGridTDF, voxelGridDimX * voxelGridDimY * voxelGridDimZ * sizeof(float), hipMemcpyHostToDevice);
	marvin::checkCUDA(__LINE__, hipGetLastError());

	int cudaLoopNum = (int)ceil((float)(voxelGridDimX * voxelGridDimY * voxelGridDimZ) / (float)(CUDA_NUM_THREADS * CUDA_MAX_NUM_BLOCKS));
	for (int i = 0; i < cudaLoopNum; ++i)
	{
		computeTDFKernel <<< CUDA_MAX_NUM_BLOCKS, CUDA_NUM_THREADS >>>(i, d_voxelGridOcc, d_voxelGridTDF,
				voxelGridDimX, voxelGridDimY, voxelGridDimZ,
				voxelSize, truncMargin);
	}

	hipMemcpy(voxelGridTDF, d_voxelGridTDF, voxelGridDimX * voxelGridDimY * voxelGridDimZ *sizeof(float), hipMemcpyDeviceToHost);
	marvin::checkCUDA(__LINE__, hipGetLastError());

	hipFree(d_voxelGridOcc);
	hipFree(d_voxelGridTDF);
}

void computeTDFVoxelGridSun(float * voxelGridOcc,
						 float * voxelGridTDF,
						 int voxelGridDimX,
						 int voxelGridDimY,
						 int voxelGridDimZ,
						 float voxelSize,
						 float truncMargin,
						 int modelNum)
{
	float * d_voxelGridOcc;
	float * d_voxelGridTDF;
	hipMalloc(&d_voxelGridOcc, modelNum * voxelGridDimX * voxelGridDimY * voxelGridDimZ * sizeof(float));
	hipMalloc(&d_voxelGridTDF, modelNum * voxelGridDimX * voxelGridDimY * voxelGridDimZ * sizeof(float));
	marvin::checkCUDA(__LINE__, hipGetLastError());
	hipMemcpy(d_voxelGridOcc, voxelGridOcc, modelNum * voxelGridDimX * voxelGridDimY * voxelGridDimZ * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_voxelGridTDF, voxelGridTDF, modelNum * voxelGridDimX * voxelGridDimY * voxelGridDimZ * sizeof(float), hipMemcpyHostToDevice);
	marvin::checkCUDA(__LINE__, hipGetLastError());

	int cudaLoopNum = (int)ceil((float)(voxelGridDimX * voxelGridDimY * voxelGridDimZ) / (float)(CUDA_NUM_THREADS * CUDA_MAX_NUM_BLOCKS));
	for (int i = 0; i < cudaLoopNum; ++i)
	{
		computeTDFKernelSun <<< CUDA_MAX_NUM_BLOCKS, CUDA_NUM_THREADS >>>(i, d_voxelGridOcc, d_voxelGridTDF,
				voxelGridDimX, voxelGridDimY, voxelGridDimZ,
				voxelSize, truncMargin, modelNum);
	}

	hipMemcpy(voxelGridTDF, d_voxelGridTDF, modelNum * voxelGridDimX * voxelGridDimY * voxelGridDimZ *sizeof(float), hipMemcpyDeviceToHost);
	marvin::checkCUDA(__LINE__, hipGetLastError());

	hipFree(d_voxelGridOcc);
	hipFree(d_voxelGridTDF);
}

void compute3DMatchDescriptor(int keypointNum,
							  int batchSize,
							  int descriptorSize,
							  float * keypointDescriptors,
							  float * keypointGrid,
							  float * voxelGridTDF,
							  int voxelGridDimX, int voxelGridDimY, int voxelGridDimZ)
{
	// Start Marvin network
	//std::string homePath(getenv("HOME"));
	//auto startTime = std::chrono::system_clock::now();
	//marvin::Net convnet(homePath + "/iroboscan/Seg/src/Cuda/3dmatch-net-test.json");
	//convnet.Malloc(marvin::Testing);
	//convnet.loadWeights(homePath + "/iroboscan/Seg/src/Cuda/3dmatch-weights-snapshot-137000.marvin");
	//auto endTime = std::chrono::system_clock::now();
	//auto duration = std::chrono::duration_cast<std::chrono::microseconds>(endTime - startTime);
	//std::cout << "Spend " << (double)duration.count() * std::chrono::microseconds::period::num / std::chrono::microseconds::period::den  << " seconds loading weights." << std::endl;
	marvin::Response * rData;
	marvin::Response * rFeat;
	rData = Marvin3DMatch::getInstance().getConvolutionalNet()->getResponse("data");
	rFeat = Marvin3DMatch::getInstance().getConvolutionalNet()->getResponse("feat");
	//std::cout << "3D Match network architecture successfully loaded into Marvin!" << std::endl;

	// Run forward passes with Marvin to get 3DMatch descriptors for each keypoint
	StorageT * batchTDF = new StorageT[batchSize * 30 * 30 * 30];
	//std::cout << "Computing 3DMatch descriptor for " << keypointNum << " keypoints..." << std::endl;
	for (int i = 0; i < (keypointNum / batchSize); ++i)
	{
		for (int j = i * batchSize; j < (i + 1) * batchSize; ++j)
		{
			int batchKeyptIDX = j - i * batchSize;
			float keyptGridX = keypointGrid[3 * j];
			float keyptGridY = keypointGrid[3 * j + 1];
			float keyptGridZ = keypointGrid[3 * j + 2];

			StorageT * localVoxelGridTDF = new StorageT[30 * 30 * 30];
			int localVoxelIDX = 0;
			for (int z = keyptGridZ - 15; z < keyptGridZ + 15; ++z)
			{
				for (int y = keyptGridY - 15; y < keyptGridY + 15; ++y)
				{
					for (int x = keyptGridX - 15; x < keyptGridX + 15; ++x)
					{
						localVoxelGridTDF[localVoxelIDX] = CPUCompute2StorageT(voxelGridTDF[z * voxelGridDimX * voxelGridDimY + y * voxelGridDimX + x]);
						localVoxelIDX++;
					}
				}
			}
			memcpy(batchTDF + batchKeyptIDX * 30 * 30 * 30, localVoxelGridTDF, sizeof(StorageT) * 30 * 30 * 30);
			//for (int k = 0; k < 30 * 30 * 30; ++k)
			//{
			//	batchTDF[batchKeyptIDX * 30 * 30 * 30 + k] = localVoxelGridTDF[k];
			//}
			delete [] localVoxelGridTDF;
		}

		// Pass local TDF patches through Marvin
		hipMemcpy(rData->dataGPU, batchTDF, rData->numBytes(), hipMemcpyHostToDevice);
		marvin::checkCUDA(__LINE__, hipGetLastError());
		Marvin3DMatch::getInstance().getConvolutionalNet()->forward();

		// Copy descriptor vectors from GPU to CPU memory
		StorageT * batchDescriptor = new StorageT[batchSize * descriptorSize];
		hipMemcpy(batchDescriptor, rFeat->dataGPU, rFeat->numBytes(), hipMemcpyDeviceToHost);
		marvin::checkCUDA(__LINE__, hipGetLastError());

		for (int j = 0; j < batchSize * descriptorSize; ++j)
		{
			keypointDescriptors[i * batchSize * descriptorSize + j] = CPUStorage2ComputeT(batchDescriptor[j]);
		}
		delete [] batchDescriptor;
	}
	delete [] batchTDF;
}

__global__ void computeCorrespondPairKernel(float * patchDescriptor,
												float * modelDescriptor,
												int modelNum,
												int patchKeypointNum,
												int modelKeypointNum,
												int descriptorSize,
												double paraMaxRank,
												double paraCoefRank,
												double paraExpRank,
												double * correspondCost)
{
	int modelKeyptId = blockIdx.x;
	int patchKeyptId = threadIdx.x;

	__shared__ float descriptors[512];


//	__shared__ float descriptors[512];

		//descriptors[keyptIndex] = patchDescriptor[i * descriptorSize + keyptIndex];
		//if (keyptIndex < 12)
		//{
			//descriptors[keyptIndex + 500] = patchDescriptor[i * descriptorSize + keyptIndex + 500];
		//}
		//__syncthreads();
	
		//for (int j = 0; j < 20; ++j)
		//{
	if (modelKeyptId < modelKeypointNum)
	{
		for (int i = 0; i < modelNum; ++i)
		{
			descriptors[patchKeyptId] = modelDescriptor[i * modelKeypointNum * descriptorSize + modelKeyptId * descriptorSize + patchKeyptId];
			__syncthreads();

			if (patchKeyptId < patchKeypointNum)
			{
				double rankCost = 0.0;
				for (int j = 0; j < descriptorSize; ++j)
				{
					rankCost += pow(descriptors[j] - patchDescriptor[j * patchKeypointNum + patchKeyptId], 2);
				}
				rankCost = sqrt(rankCost);
				if (rankCost > paraMaxRank)
				{
					correspondCost[i * modelKeypointNum * patchKeypointNum + modelKeyptId * patchKeypointNum + patchKeyptId] = 999999;
				}
				else
				{
					correspondCost[i * modelKeypointNum * patchKeypointNum + modelKeyptId * patchKeypointNum + patchKeyptId] = paraCoefRank * pow(rankCost, paraExpRank);
				}
			}
			__syncthreads();
		}
	}
		//}
		//__syncthreads();
}

void computeCorrespondPair(float * patchDescriptor,
							   float * modelDescriptor,
							   int modelNum,
							   int patchKeypointNum,
							   int modelKeypointNum,
							   int descriptorSize,
							   double paraMaxRank,
							   double paraCoefRank,
							   double paraExpRank,
							   double * correspondCost)
{
	float * d_patchDescriptor;
	float * d_modelDescriptor;
	double * d_correspondCost;

	hipMalloc(&d_patchDescriptor, patchKeypointNum * descriptorSize * sizeof(float));
	hipMalloc(&d_modelDescriptor, modelNum * modelKeypointNum * descriptorSize * sizeof(float));
	hipMalloc(&d_correspondCost, modelNum * patchKeypointNum * modelKeypointNum * sizeof(double));

	float * patchDescriptorTrans = new float[patchKeypointNum * descriptorSize];
	for (int i = 0; i < patchKeypointNum; ++i)
	{
		for (int j = 0; j < descriptorSize; ++j)
		{
			patchDescriptorTrans[j * patchKeypointNum + i] = patchDescriptor[i * descriptorSize + j];
		}
	}

	hipMemcpy(d_patchDescriptor, patchDescriptorTrans, patchKeypointNum * descriptorSize * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_modelDescriptor, modelDescriptor, modelNum * modelKeypointNum * descriptorSize * sizeof(float), hipMemcpyHostToDevice);

	int threadsPerBlock = 512;
	int blocksPerGrid = 512;


	//computeCorrespondPairKernel <<< blocksPerGrid, threadsPerBlock, 20 * descriptorSize * sizeof(float)>>>(d_patchDescriptor, d_modelDescriptor, 
			//patchKeypointNum, modelKeypointNum, descriptorSize, paraMaxRank, paraCoefRank, paraExpRank, d_correspondCost);
	hipEvent_t start1;
	hipEventCreate(&start1);
	hipEvent_t stop1;
	hipEventCreate(&stop1);
	hipEventRecord(start1, NULL);
	computeCorrespondPairKernel <<< blocksPerGrid, threadsPerBlock>>>(d_patchDescriptor, d_modelDescriptor, modelNum, 
			patchKeypointNum, modelKeypointNum, descriptorSize, paraMaxRank, paraCoefRank, paraExpRank, d_correspondCost);
	hipEventRecord(stop1, NULL);
	hipEventSynchronize(stop1);
	float mescTotal1 = 0.0f;
	hipEventElapsedTime(&mescTotal1, start1, stop1);
	std::cout << mescTotal1 << " msec" << std::endl;

	hipMemcpy(correspondCost, d_correspondCost, modelNum * patchKeypointNum * modelKeypointNum * sizeof(double), hipMemcpyDeviceToHost);

	hipFree(d_patchDescriptor);
	hipFree(d_modelDescriptor);
	hipFree(d_correspondCost);
	delete [] patchDescriptorTrans;
}

__global__ void labelPointToClosestSeedKernel(int loopIndex, int pointNum, int keypointNum, float * pointPositions, int * seedIndexes, int * pointLabels, int * neighborIndexes)
{
	int pointIndex = loopIndex * CUDA_NUM_THREADS * CUDA_MAX_NUM_BLOCKS + blockIdx.x * CUDA_NUM_THREADS + threadIdx.x;

	if (pointIndex >= pointNum)
		return;

	float posX = pointPositions[3 * pointIndex];
	float posY = pointPositions[3 * pointIndex + 1];
	float posZ = pointPositions[3 * pointIndex + 2];

	int closestSeedIndex = -1;
	int neighborIndex = -1;
	float minDistance = 99999;
	float neighborDistance = 99999;
	for (int i = 0; i < keypointNum; ++i)
	{
		int seedIndex = seedIndexes[i];
		float distance = sqrt(pow(posX - pointPositions[3 * seedIndex], 2) + pow(posY - pointPositions[3 * seedIndex + 1], 2) + pow(posZ - pointPositions[3 * seedIndex + 2], 2));
		if (distance < minDistance)
		{
			neighborIndex = closestSeedIndex;
			neighborDistance = minDistance;
			closestSeedIndex = i;
			minDistance = distance;
		}
		else if (distance < neighborDistance)
		{
			neighborIndex = i;
			neighborDistance = distance;
		}
	}
	pointLabels[pointIndex] = closestSeedIndex;
	neighborIndexes[pointIndex] = neighborIndex;
}

void labelPointToClosestSeed(int pointNum,
							 int keypointNum,
							 const float * pointPositions,
							 const int * seedIndexes,
							 int * pointLabels,
							 int * neighborIndexes)
{
	float * d_pointPositions;
	int * d_seedIndexes;
	int * d_pointLabels;
	int * d_neighborIndexes;

	hipMalloc(&d_pointPositions, pointNum * 3 * sizeof(float));
	hipMalloc(&d_seedIndexes, keypointNum * sizeof(int));
	hipMalloc(&d_pointLabels, pointNum * sizeof(int));
	hipMalloc(&d_neighborIndexes, pointNum * sizeof(int));

	hipMemcpy(d_pointPositions, pointPositions, pointNum * 3 * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_seedIndexes, seedIndexes, keypointNum * sizeof(int), hipMemcpyHostToDevice);

	int cudaLoopNum = (int)ceil((float)(pointNum) / (float)(CUDA_NUM_THREADS * CUDA_MAX_NUM_BLOCKS));
	for (int i = 0; i < cudaLoopNum; ++i)
	{
		labelPointToClosestSeedKernel <<< CUDA_MAX_NUM_BLOCKS, CUDA_NUM_THREADS >>>(i, pointNum, keypointNum, d_pointPositions, d_seedIndexes, d_pointLabels, d_neighborIndexes);
	}

	hipMemcpy(pointLabels, d_pointLabels, pointNum * sizeof(int), hipMemcpyDeviceToHost);
	hipMemcpy(neighborIndexes, d_neighborIndexes, pointNum * sizeof(int), hipMemcpyDeviceToHost);

	hipFree(d_pointPositions);
	hipFree(d_seedIndexes);
	hipFree(d_pointLabels);
	hipFree(d_neighborIndexes);
}
